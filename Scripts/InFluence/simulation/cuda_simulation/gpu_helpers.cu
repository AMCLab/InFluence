#include "hip/hip_runtime.h"
#include "gpu_helpers.cuh"
#include <cmath>


__device__ int custom_round_down(double x, int decimals) {
    double multiplier = pow(10.0, static_cast<double>(decimals));
    double rounded_value = ceil(fabs(x) * multiplier - 0.5);
    rounded_value /= multiplier;
    return static_cast<int>(rounded_value);
}

__device__ double evaluate_alpha(double E, double AlphaMultiplier) {
    return AlphaMultiplier / E;
}

__device__ double evaluate_path_length(double CrossSection, double PathLengthMultiplier) {
    double PathLength = PathLengthMultiplier * (1.0 / CrossSection);
    return PathLength;
}

__device__ float evaluate_cross_section_opt(float E, float CrossSectionLogArgMultiplier, float CrossSectionNumorator, float CrossSectionDenominatorA, float CrossSectionDenominatorB) {
    float LogArg = log10f(E * CrossSectionLogArgMultiplier);
    float CrossSection = CrossSectionNumorator / ((E + sqrtf(E) * CrossSectionDenominatorA) * (1.0f - expf(-1.0f * LogArg * LogArg) * CrossSectionDenominatorB));
    return CrossSection;
}

__device__ double evaluate_step(double path_length, double RandomStep) {
    return -path_length * logf(RandomStep);
}

__device__ double evaluate_phi(double RandomNum, double alpha) {
    double cosPhi = 1.0 - (2.0 * alpha * pow(RandomNum, 2)) / (1.0 + alpha - RandomNum);
    return acosf(cosPhi);
}

__device__ double evaluate_pho(double RandomNum, double pi) {
    return 2.0 * pi * RandomNum;
}


__device__ double evaluate_direction_cosine_a(double phi, double psi, double cosineY, double cosineZ) {
    float alpha = sinf(psi) * sinf(phi);
    float beta = sinf(phi) * cosf(psi);
    float gamma = cosf(phi);
    float cos_1 = static_cast<float>(cosineZ);
    float sin_1 = sqrtf(1.0f - cos_1 * cos_1);
    float cos_2 = static_cast<float>(cosineY) / sin_1;
    float sin_2 = sqrtf(1.0f - cos_2 * cos_2);
    return static_cast<double>(alpha * cos_2 + sin_2 * (beta * cos_1 + gamma * sin_1));
}


__device__ double evaluate_direction_cosine_c(double phi, double psi, double cosineZ) {
    float beta = sinf(static_cast<float>(phi)) * cosf(static_cast<float>(psi));
    float gamma = cosf(static_cast<float>(phi));
    float sin_1 = sqrtf(1.0f - static_cast<float>(cosineZ) * static_cast<float>(cosineZ));
    float cos_1 = static_cast<float>(cosineZ);

    return static_cast<double>(-beta * sin_1 + gamma * cos_1);
}

__device__ double evaluate_energy_loss_rate(double E, double EnergyLossMultiplierA, double EnergyLossMultiplier) {
    float log_arg = logf(1.166f * (E + 0.85f * static_cast<float>(EnergyLossMultiplier)) / static_cast<float>(EnergyLossMultiplier));
    return static_cast<double>(EnergyLossMultiplierA * (1.0f / static_cast<float>(E)) * log_arg);
}

__device__ double get_random_uniform(double a, double b) {
    hiprandState state;
    hiprand_init(clock64(), 0, 0, &state); // Seed the random number generator

    double random_value = hiprand_uniform_double(&state); // Generate a random double between 0 and 1
    return a + random_value * (b - a); // Scale to [a, b]
}

__device__ void initialize_positions(double step, double ProbeDiameter, double* position_data) {
    double z0 = 1e-2 + step;
    double x0 = ProbeDiameter * get_random_uniform(-1.0, 1.0);
    double y0 = ProbeDiameter * get_random_uniform(-1.0, 1.0);
    double vector_length = sqrtf(x0 * x0 + y0 * y0 + z0 * z0);
    double cosineX = x0 / vector_length;
    double cosineY = y0 / vector_length;
    double cosineZ = z0 / vector_length;

    // Store the results in the custom data structure (array)
    position_data[0] = cosineX;
    position_data[1] = cosineY;
    position_data[2] = cosineZ;
    position_data[3] = z0;
    position_data[4] = y0;
    position_data[5] = x0;
    position_data[6] = vector_length;
}