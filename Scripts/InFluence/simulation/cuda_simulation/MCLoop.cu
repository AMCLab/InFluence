#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>




__device__ float getRand(float a, float b, hiprandState* state)
{
    return a + (b - a) * hiprand_uniform(state);
}


extern "C" {


__device__ int custom_round_down(double x, int decimals=0) {
    double multiplier = pow(10.0, static_cast<double>(decimals));
    double rounded_value = ceil(fabs(x) * multiplier - 0.5);
    rounded_value /= multiplier;
    return static_cast<int>(rounded_value);
}

__device__ double evaluate_alpha(double E, double AlphaMultiplier) {
    return AlphaMultiplier / E;
}

__device__ double evaluate_path_length(double CrossSection, double PathLengthMultiplier) {
    double PathLength = PathLengthMultiplier * (1.0 / CrossSection);
    return PathLength;
}

__device__ float evaluate_cross_section_opt(float E, float CrossSectionLogArgMultiplier, float CrossSectionNumorator, float CrossSectionDenominatorA, float CrossSectionDenominatorB) {
    float LogArg = log10f(E * CrossSectionLogArgMultiplier);
    float CrossSection = CrossSectionNumorator / ((E + sqrtf(E) * CrossSectionDenominatorA) * (1.0f - expf(-1.0f * LogArg * LogArg) * CrossSectionDenominatorB));
    return CrossSection;
}

__device__ double evaluate_step(double path_length, double RandomStep) {
    return -path_length * logf(RandomStep);
}

__device__ double evaluate_phi(double RandomNum, double alpha) {
    double cosPhi = 1.0 - (2.0 * alpha * pow(RandomNum, 2)) / (1.0 + alpha - RandomNum);
    return acosf(cosPhi);
}

__device__ double evaluate_pho(double RandomNum, double pi) {
    return 2.0 * pi * RandomNum;
}
__device__ double evaluate_direction_cosine_a(double phi, double psi, double cosineX, double cosineY, double cosineZ) {
    float alpha = sinf(psi) * sinf(phi);
    float beta = sinf(phi) * cosf(psi);
    float gamma = cosf(phi);
    float cos_1 = static_cast<float>(cosineZ);
    float sin_1 = sqrtf(1.0f - cos_1 * cos_1);
    float cos_2 = static_cast<float>(cosineY) / sin_1;
    float sin_2 = static_cast<float>(cosineX) / sin_1;  // Corrected line
    return static_cast<double>(alpha * cos_2 + sin_2 * (beta * cos_1 + gamma * sin_1));
}

__device__ double evaluate_direction_cosine_b(double phi, double psi, double cosineX, double cosineY, double cosineZ) {
    float alpha = sinf(static_cast<float>(phi)) * sinf(static_cast<float>(psi));
    float beta = sinf(static_cast<float>(phi)) * cosf(static_cast<float>(psi));
    float gamma = cosf(static_cast<float>(phi));
    float sin_1 = sqrtf(1.0f - static_cast<float>(cosineZ) * static_cast<float>(cosineZ));
    float cos_2 = static_cast<float>(cosineY) / sin_1;
    float sin_2 = static_cast<float>(cosineX) / sin_1;

    return static_cast<double>(-alpha * sin_2 + cos_2 * (beta * static_cast<float>(cosineZ) + gamma * sin_1));
}


__device__ double evaluate_direction_cosine_c(double phi, double psi, double cosineZ) {
    float beta = sinf(static_cast<float>(phi)) * cosf(static_cast<float>(psi));
    float gamma = cosf(static_cast<float>(phi));
    float sin_1 = sqrtf(1.0f - static_cast<float>(cosineZ) * static_cast<float>(cosineZ));
    float cos_1 = static_cast<float>(cosineZ);

    return static_cast<double>(-beta * sin_1 + gamma * cos_1);
}

__device__ double evaluate_energy_loss_rate(double E, double EnergyLossMultiplierA, double EnergyLossMultiplier) {
    float log_arg = logf(1.166f * (E + 0.85f * static_cast<float>(EnergyLossMultiplier)) / static_cast<float>(EnergyLossMultiplier));
    return static_cast<double>(EnergyLossMultiplierA * (1.0f / static_cast<float>(E)) * log_arg);
}


__device__ void initialize_positions(double step, double ProbeDiameter, double* position_data, hiprandState* state) {
    double z0 = 1e-2 + step;
    double x0 = ProbeDiameter * getRand(-1.0, 1.0, state);
    double y0 = ProbeDiameter * getRand(-1.0, 1.0, state);
    double vector_length = sqrtf(x0 * x0 + y0 * y0 + z0 * z0);
    double cosineX = x0 / vector_length;
    double cosineY = y0 / vector_length;
    double cosineZ = z0 / vector_length;

    // Store the results in the custom data structure (array)
    position_data[0] = cosineX;
    position_data[1] = cosineY;
    position_data[2] = cosineZ;
    position_data[3] = z0;
    position_data[4] = y0;
    position_data[5] = x0;
    position_data[6] = vector_length;
}



__device__ void atomicAdd_double(double* address, double value) {
        unsigned long long int* address_as_ull = (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(value +
                            __longlong_as_double(assumed)));
        } while (assumed != old);
    }


__global__ void MCScatteringSimulationKernel(
    const double* pixels, 
    int numPixels, 
    double E_i, 
    double ProbeDiameter,
    double MinimumEnergy, 
    double dE_threshold, 
    int perfect_image_0,
    int perfect_image_1, 
    double Density, 
    double t_counting,
    double AlphaMultiplier, 
    double CrossSectionNumorator, 
    double CrossSectionLogArgMultiplier, 
    double CrossSectionDenominatorA, 
    double CrossSectionDenominatorB, 
    double PathLengthMultiplier, 
    double EnergyLossMultiplierA, 
    double EnergyLossMultiplierB,
    double* new_image_MCS) 
                                                
    {
    
    int threadIndex_x = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex_y = blockIdx.y * blockDim.y + threadIdx.y;   

    

    // Convert 2D thread indices to 1D index
    int threadIndex = threadIndex_y * perfect_image_1 + threadIndex_x;

    hiprandState state;
    hiprand_init(1234, threadIndex, 0, &state); 
    

    // Check if the thread index is within the valid range of pixels
    if (threadIndex < numPixels) {
        // Get pixel data for this thread
        int pixelOffset = threadIndex * 6; // Each pixel has 6 values: count, i_coordinate, j_coordinate, x_dimension, y_dimension, z_dimension
        int count = static_cast<int>(pixels[pixelOffset]);
        double i_coordinate = pixels[pixelOffset + 1];
        double j_coordinate = pixels[pixelOffset + 2];
        double x_dimension = pixels[pixelOffset + 3];
        double y_dimension = pixels[pixelOffset + 4];
        double z_dimension = pixels[pixelOffset + 5];


        // Initialize 2D arrays as device arrays (raw pointers)
        int pixelIndex = static_cast<int>(i_coordinate) * perfect_image_0 + static_cast<int>(j_coordinate);
        //eh_charge_counter[pixelIndex] = 0.0;
        //new_image_MCS[pixelIndex] = 0.0;

        int number_transmitted = 0;
        int number_eh_pairs = 0;
        int number_stopped  = 0;
        int number_backscattered = 0;


        if (count == 0) {
            // Do nothing for this pixel

        } else {
            // Loop 'count' times for this pixel



            for (int k = 0; k < count; ++k) {

                const int max_nnz = 50;  // Maximum number of non-zero values
                double values[max_nnz];
                int indices[max_nnz];
                int nnz = 0;  // This will keep track of the number of non-zero values currently stored
                int localCount = 0;
                // Initialize variables for electron conditions
                double alpha = evaluate_alpha(E_i, AlphaMultiplier);
                double CrossSection = evaluate_cross_section_opt(E_i, CrossSectionLogArgMultiplier, CrossSectionNumorator, CrossSectionDenominatorA, CrossSectionDenominatorB);
                double PathLength = evaluate_path_length(CrossSection, PathLengthMultiplier);
                double RND_step = getRand(0.000001, 0.999999, &state);
                double step = evaluate_step(PathLength, RND_step);  

                // Initialize position data
                double position_data[7];
                initialize_positions(step, ProbeDiameter, position_data, &state);
                double cosineX = position_data[0];
                double cosineY = position_data[1];
                double cosineZ = position_data[2];
                double z0 = position_data[3];
                double y0 = position_data[4];
                double x0 = position_data[5];
                double E = E_i;
                bool condition = true;


                while (condition == true) {
                
                
                // Generate random numbers
                double RND_phi = getRand(0, 1, &state);
                double RND_step = getRand(0.000001, 0.999999, &state);
                double RND_pho = getRand(0, 1, &state);

                // Calculate alpha, cross section, path length, and step
                double alpha = evaluate_alpha(E_i, AlphaMultiplier);
                double CrossSection = evaluate_cross_section_opt(E, CrossSectionLogArgMultiplier, CrossSectionNumorator, CrossSectionDenominatorA, CrossSectionDenominatorB);
                double PathLength = evaluate_path_length(CrossSection, PathLengthMultiplier);
                double step = evaluate_step(PathLength, RND_step);

                // Calculate energy loss and update energy
                double dE = step * Density * evaluate_energy_loss_rate(E, EnergyLossMultiplierA, EnergyLossMultiplierB);
                E = E + dE;

                // Calculate scattering angles and direction cosines
                double phi = evaluate_phi(RND_phi, alpha);
                double psi = evaluate_pho(RND_pho, M_PI); 

                double ca = evaluate_direction_cosine_a(phi, psi, cosineX, cosineY, cosineZ);
                double cb = evaluate_direction_cosine_b(phi, psi, cosineX, cosineY, cosineZ);
                double cc = evaluate_direction_cosine_c(phi, psi, cosineZ);

                // Update positions
                x0 = x0 + step * ca;
                y0 = y0 + step * cb;
                z0 = z0 + step * cc;

                // Update direction cosines
                cosineX = ca;
                cosineY = cb;
                cosineZ = cc;

                
                ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                //      Scenario 1 : Electron stops in material.                                                                 //
                //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                if (E <= MinimumEnergy) {
                    number_stopped++;
                    condition = false; 

                }

                ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                //      Scenario 2 : Electron backscatters.                                                                     //
                //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                if (z0 < std::pow(10, -2)) {
                    number_backscattered++;
                    condition = false; 
                    
                }

                ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                //      Scenario 3 : Electron penetrates the material.                                                          //
                //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                if (z0 > z_dimension) {
                    number_transmitted++;
                    condition = false;
                   
                }

                ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                //      Scenario 4 : Electron deposits sufficient energy.                                                        //
                //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                if (-dE >= dE_threshold) {
                    int new_eh_pairs = std::floor(-dE / dE_threshold);
                    number_eh_pairs += new_eh_pairs;
                    

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 5 : Electron deposits sufficient energy and stays within pixel boundary.               //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    if ((x0 <= x_dimension) && (x0 >= -x_dimension) &&
                        (y0 <= y_dimension) && (y0 >= -y_dimension)) {
                        int pixel_1D_index = static_cast<int>(i_coordinate) * perfect_image_1 + static_cast<int>(j_coordinate);
                        
                        bool found = false;
                        for (int i = 0; i < nnz; ++i) {
                            if (indices[i] == pixel_1D_index) {
                                values[i] += static_cast<double>(new_eh_pairs);
                                found = true;
                                break;
                            }
                        }
                        
                        if (!found && nnz < max_nnz) {
                            indices[nnz] = pixel_1D_index;
                            values[nnz] = static_cast<double>(new_eh_pairs);
                            nnz++;
                        }

                    }

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 6 : Electron deposits sufficient energy and moves positively in x and y.                       //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    else if ((x0 > x_dimension) && (y0 > y_dimension)) {
                        
                        // Electron moves positively in x and y

                        

                        // Calculate the translations
                        int translation_x = custom_round_down(x0 / (2 * x_dimension));
                        int translation_y = custom_round_down(y0 / (2 * y_dimension));

                        

                        if ((i_coordinate + translation_x <= perfect_image_0 - 1) && (j_coordinate + translation_y <= perfect_image_1 - 1)) {
                            // Update the value in the device array using an atomic add
                            
                            int pixel_1D_index = (i_coordinate + translation_x) * perfect_image_1 + (j_coordinate + translation_y);
                            

                            bool found = false;
                            for (int i = 0; i < nnz; ++i) {
                                if (indices[i] == pixel_1D_index) {
                                    values[i] += static_cast<double>(new_eh_pairs);
                                    found = true;
                                    break;
                                }
                            }
                            
                            if (!found && nnz < max_nnz) {
                                indices[nnz] = pixel_1D_index;
                                values[nnz] = static_cast<double>(new_eh_pairs);
                                nnz++;
                            }

                        }
                    }                 

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 7 : Electron deposits sufficient energy and moves negatively in x and y.                       //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    else if ((x0 < -x_dimension) && (y0 < -y_dimension)) {


                        // Calculate the translations
                        int translation_x = custom_round_down(x0 / (2 * x_dimension));
                        int translation_y = custom_round_down(y0 / (2 * y_dimension));

                        if ((i_coordinate - translation_x >= 0) && (j_coordinate - translation_y >= 0)) {

                        
                        int pixel_1D_index = (i_coordinate - translation_x) * perfect_image_1 + (j_coordinate - translation_y);

                            bool found = false;
                            for (int i = 0; i < nnz; ++i) {
                                if (indices[i] == pixel_1D_index) {
                                    values[i] += static_cast<double>(new_eh_pairs);
                                    found = true;
                                    break;
                                }
                            }
                            
                            if (!found && nnz < max_nnz) {
                                indices[nnz] = pixel_1D_index;
                                values[nnz] = static_cast<double>(new_eh_pairs);
                                nnz++;
                            }

                        }
                    }

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 8 : Electron deposits sufficient energy and moves positively in x only.                        //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    else if (x0 > x_dimension) {
                        int translation_x = custom_round_down(x0 / (2 * x_dimension));
                        if (i_coordinate + translation_x <= perfect_image_0 - 1) {
                        // Update the value in the device array using an atomic add
                        
                        int pixel_1D_index = (i_coordinate + translation_x) * perfect_image_1 + j_coordinate;
                        
                            bool found = false;
                            for (int i = 0; i < nnz; ++i) {
                                if (indices[i] == pixel_1D_index) {
                                    values[i] += static_cast<double>(new_eh_pairs);
                                    found = true;
                                    break;
                                }
                            }
                            
                            if (!found && nnz < max_nnz) {
                                indices[nnz] = pixel_1D_index;
                                values[nnz] = static_cast<double>(new_eh_pairs);
                                nnz++;
                            }

                        }
                    }

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 9 : Electron deposits sufficient energy and moves negatively in x only.                        //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    else if (x0 < -x_dimension) {
                        int translation_x = custom_round_down(x0 / (2 * x_dimension));
                        if (i_coordinate - translation_x >= 0) {
                        // Update the value in the device array using an atomic add
                        
                        int pixel_1D_index = (i_coordinate - translation_x) * perfect_image_1 + j_coordinate;

                            bool found = false;
                            for (int i = 0; i < nnz; ++i) {
                                if (indices[i] == pixel_1D_index) {
                                    values[i] += static_cast<double>(new_eh_pairs);
                                    found = true;
                                    break;
                                }
                            }
                            
                            if (!found && nnz < max_nnz) {
                                indices[nnz] = pixel_1D_index;
                                values[nnz] = static_cast<double>(new_eh_pairs);
                                nnz++;
                            }

                        }
                        }

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 10 : Electron deposits sufficient energy and moves positively in y only.                       //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    else if (y0 > y_dimension) {
                        int translation_y = custom_round_down(y0 / (2 * y_dimension));
                        if (j_coordinate + translation_y <= perfect_image_1 - 1) {
                        // Update the value in the device array using an atomic add
                        
                        int pixel_1D_index = i_coordinate * perfect_image_1 + (j_coordinate + translation_y);
                        

                        bool found = false;
                        for (int i = 0; i < nnz; ++i) {
                            if (indices[i] == pixel_1D_index) {
                                values[i] += static_cast<double>(new_eh_pairs);
                                found = true;
                                break;
                            }
                        }
                            
                            if (!found && nnz < max_nnz) {
                                indices[nnz] = pixel_1D_index;
                                values[nnz] = static_cast<double>(new_eh_pairs);
                                nnz++;
                            }

                        }    
                           
                        }
                    

                    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //      Scenario 11 : Electron deposits sufficient energy and moves negatively in y only.                       //
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

                    else if (y0 < -y_dimension) {
                        int translation_y = custom_round_down(y0 / (2 * y_dimension));
                        if (j_coordinate - translation_y >= 0) {
                        // Update the value in the device array using an atomic add
                        
                        int pixel_1D_index = i_coordinate * perfect_image_1 + (j_coordinate - translation_y);
                        

                        bool found = false;
                        for (int i = 0; i < nnz; ++i) {
                            if (indices[i] == pixel_1D_index) {
                                values[i] += static_cast<double>(new_eh_pairs);
                                found = true;
                                break;
                            }
                        }
                            
                            if (!found && nnz < max_nnz) {
                                indices[nnz] = pixel_1D_index;
                                values[nnz] = static_cast<double>(new_eh_pairs);
                                nnz++;
                            }


                        }
                    }
                
                }  // Sufficient energy loop
            } // wile loop

            for (int i = 0; i < nnz; ++i) { // nnz is the number of non-zero entries 
                int globalIndex = indices[i];
                values[i] = floor(dE_threshold * values[i] / t_counting);
                atomicAdd_double(&(new_image_MCS[globalIndex]), values[i]);
                
            }
     

            } //electron level
        
        } //else

    } //pixel_lop


}

} // c linkage


